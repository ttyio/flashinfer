#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "flashinfer/gemm/fp4_gemm_template.h"

namespace flashinfer {
namespace gemm {
template class CutlassFp4GemmRunner<__hip_bfloat16, FP4GemmType::W4A4_NVFP4_NVFP4>;
template class CutlassFp4GemmRunner<half, FP4GemmType::W4A4_NVFP4_NVFP4>;
}  // namespace gemm
}  // namespace flashinfer
